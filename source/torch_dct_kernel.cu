#include "hip/hip_runtime.h"

#include <stdio.h>

#include "../include/utils.h"

template <typename scalar_t>
__global__ void cudaNaiveDCT2DKernel(const uint numTotalThreads, const uint batchSizeDim, const uint channelDim, const uint heightDim, const uint widthDim, const scalar_t* __restrict__ input, const uint numPoints, scalar_t* __restrict__ output) {
    
    const uint idx = threadIdx.x + blockIdx.x * blockDim.x
                   + (threadIdx.y + blockIdx.y * blockDim.y) * gridDim.x * blockDim.x
                   + (threadIdx.z + blockIdx.z * blockDim.z) * gridDim.x * gridDim.y * blockDim.x * blockDim.y;

    if (idx < numTotalThreads) {
        
        const uint chwDim = channelDim * heightDim * widthDim;
        const uint hwDim = heightDim * widthDim;
        const uint n = int(idx / chwDim);
        const uint c = int((idx % chwDim) / hwDim);
        const uint h = int((idx % hwDim) / widthDim);
        const uint w = idx % widthDim;
        const uint numPointsPow2 = numPoints * numPoints;

        const float sqrt_1_2 = sqrtf(1.0f / 2);
        for (uint k = 0; k < numPoints; k++) {
            uint hk = h * numPoints + k;
            float lambda_k = k == 0 ? sqrt_1_2 : 1.0f;

            for (uint v = 0; v < numPoints; v++) {
                uint wv = w * numPoints + v;
                float lambda_v = v == 0 ? sqrt_1_2 : 1.0f;

                uint spectralIdx = n * chwDim * numPointsPow2 
                                 + c * hwDim * numPointsPow2 
                                 + hk * widthDim * numPoints 
                                 + wv; 

                for (uint i = 0; i < numPoints; i++) {
                    uint hi = h * numPoints + i;
                    float cos_i_k = cosf((2.0f * i + 1.0f) * k * M_PI / (2.0f * numPoints));

                    for (uint j = 0; j < numPoints; j++) {
                        uint wj = w * numPoints + j;
                        float cos_j_v = cosf((2.0f * j + 1.0f) * v * M_PI / (2.0f * numPoints));
                        
                        uint specialIdx = n * chwDim * numPointsPow2 
                                        + c * hwDim * numPointsPow2 
                                        + hi * widthDim * numPoints 
                                        + wj; 

                        output[spectralIdx] += input[specialIdx] * (2.0f / numPoints) * lambda_k * lambda_v * cos_i_k * cos_j_v;                    
                    }
                }
            }
        }
    }

    __syncthreads();
}

template <typename scalar_t>
__global__ void cudaNaiveIDCT2DKernel(const uint numTotalThreads, const uint batchSizeDim, const uint channelDim, const uint heightDim, const uint widthDim, const scalar_t* __restrict__ input, const uint numPoints, scalar_t* __restrict__ output) {
    
    const uint idx = threadIdx.x + blockIdx.x * blockDim.x
                   + (threadIdx.y + blockIdx.y * blockDim.y) * gridDim.x * blockDim.x
                   + (threadIdx.z + blockIdx.z * blockDim.z) * gridDim.x * gridDim.y * blockDim.x * blockDim.y;

    if (idx < numTotalThreads) {
        
        const uint chwDim = channelDim * heightDim * widthDim;
        const uint hwDim = heightDim * widthDim;
        const uint n = int(idx / chwDim);
        const uint c = int((idx % chwDim) / hwDim);
        const uint h = int((idx % hwDim) / widthDim);
        const uint w = idx % widthDim;
        const uint numPointsPow2 = numPoints * numPoints;

        const float sqrt_1_2 = sqrtf(1.0f / 2);
        for (uint i = 0; i < numPoints; i++) {
            uint hi = h * numPoints + i;

            for (uint j = 0; j < numPoints; j++) {
                uint wj = w * numPoints + j;
                
                uint specialIdx = n * chwDim * numPointsPow2 
                                + c * hwDim * numPointsPow2 
                                + hi * widthDim * numPoints 
                                + wj; 

                for (uint k = 0; k < numPoints; k++) {
                    uint hk = h * numPoints + k;
                    float lambda_k = k == 0 ? sqrt_1_2 : 1.0f;    
                    float cos_i_k = cosf((2.0f * i + 1.0f) * k * M_PI / (2.0f * numPoints));

                    for (uint v = 0; v < numPoints; v++) {
                        uint wv = w * numPoints + v;
                        float lambda_v = v == 0 ? sqrt_1_2 : 1.0f;    
                        float cos_j_v = cosf((2.0f * j + 1.0f) * v * M_PI / (2.0f * numPoints));

                        uint spectralIdx = n * chwDim * numPointsPow2 
                                         + c * hwDim * numPointsPow2 
                                         + hk * widthDim * numPoints 
                                         + wv; 

                        output[specialIdx] += input[spectralIdx] * (2.0f / numPoints) * lambda_k * lambda_v * cos_i_k * cos_j_v;
                    }
                }
            }
        }
    }

    __syncthreads();
}

at::Tensor cudaNaiveDCT2D(const at::Tensor input, const uint numPoints) {
    at::IntList inputSize = input.sizes();
    int batchSize = inputSize[0];
    int channel = inputSize[1];
    int height = inputSize[2];
    int width = inputSize[3];

    at::Tensor output = at::zeros_like(input);

    dim3 numBlocks;
    dim3 threadsPerBlock;

    uint numTotalThreads = batchSize * channel * height * width / (numPoints * numPoints);
    optimalCUDABlocksAndThreadsPerBlock(numTotalThreads, numBlocks, threadsPerBlock);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "cudaNaiveDCT2D", ([&] {
                cudaNaiveDCT2DKernel<scalar_t><<<numBlocks, threadsPerBlock>>>(
                    numTotalThreads, batchSize, channel, height / numPoints, width / numPoints, input.data_ptr<scalar_t>(), numPoints, output.data_ptr<scalar_t>()
                );
            }
        )
    );

    return output;
}

at::Tensor cudaNaiveIDCT2D(const at::Tensor input, const uint numPoints) {
    at::IntList inputSize = input.sizes();
    int batchSize = inputSize[0];
    int channel = inputSize[1];
    int height = inputSize[2];
    int width = inputSize[3];

    at::Tensor output = at::zeros_like(input);

    dim3 numBlocks;
    dim3 threadsPerBlock;

    uint numTotalThreads = batchSize * channel * height * width / (numPoints * numPoints);
    optimalCUDABlocksAndThreadsPerBlock(numTotalThreads, numBlocks, threadsPerBlock);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "cudaNaiveIDCT2D", ([&] {
                cudaNaiveIDCT2DKernel<scalar_t><<<numBlocks, threadsPerBlock>>>(
                    numTotalThreads, batchSize, channel, height / numPoints, width / numPoints, input.data_ptr<scalar_t>(), numPoints, output.data_ptr<scalar_t>()
                );
            }
        )
    );

    return output;
}
